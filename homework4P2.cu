//Problem 2. 
//Write a CUDA program: In Problem 1, change the array size to 1024

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = index; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework 4\n Problem 2: Change the array size to 1024 for Problem1 ");

//Declare int array
int size = 1024;
int *array;
int gpuThread = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int blocks = (size + gpuThread - 1) / gpuThread;
initialize<<<blocks, gpuThread>>>(array, size);

print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}
