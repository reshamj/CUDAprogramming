//Homework 5 Problem 1
//� GPU Programming: Matrix Squaring (Version 1)� or �GPU: Programming Matrix Squaring (Version 2)�.
//Team : Resham Jhangiani, Phillip Ly
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void  square ( unsigned int *matrix,  unsigned int *result,    unsigned int matrixsize) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned jj = 0; jj < matrixsize; ++jj) {
	
    for (unsigned kk = 0; kk < matrixsize; ++kk) 
    {
        result[id * matrixsize + jj] += matrix[id * matrixsize + kk] * matrix[kk * matrixsize + jj];
        	
    }
  }
}

void print(unsigned int *result, unsigned matrixsize)
{
printf("%u\n", *result);	
}

int main(void)
{
 int N=64;
 unsigned int *matrix;
 unsigned int *result;
 unsigned int matrixsize= N * sizeof(unsigned);

hipMallocManaged(&matrix, matrixsize);
//printf("%u", *matrix);
square<<<1, N>>>(matrix, result, N);// N = 64

print(result, matrixsize);
hipDeviceSynchronize();
hipDeviceReset();
return 0;
}