//Problem 3. 
//Write a CUDA program:  In Problem 2, change the array size to 8000. Check if answer to problem 3 still works.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = index; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

//Kernel function to add i to a[i]
__global__
void addIValue(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = index; i < size; i+= increment){ 
         arr[i] += i; 
      }
}


void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework 4\n Problem 4:In Problem 2, change the array size to 8000. Check if answer to problem 3 still works");

//Declare int array
int size = 8000;
int *array;
int gpuThread = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int blocks = (size + gpuThread - 1) / gpuThread;
initialize<<<blocks, gpuThread>>>(array, size);

//add value of i to array 
addIValue<<<blocks, gpuThread>>>(array, size);
hipDeviceSynchronize();

print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}
