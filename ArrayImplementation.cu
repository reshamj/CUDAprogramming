#include "hip/hip_runtime.h"
//High Performance Computing for Data Science CUDA Programming
//Experimenting with arrays

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Initialization function
__global__
void (int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = index; i < size; i += increment){
           arr[i] = 0;
        }
}

// Function to add i to array[i]
__global__
void addIValue(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = index; i < size; i+= increment){
         arr[i] += i;
      }
}

// Function to print out array for verifying correctness
void printArray(int *ar, int size){
  printf("\n");
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
}

int main(){

  int size = 32;
  int *array;
  int num_threads = 32;
  hipMallocManaged(&array, size * sizeof(int));

  // Problem 1: Initialize an array of size 32 to all zeroes
  // in parallel

  int num_blocks = (size + num_threads - 1) / num_threads;
  initialize<<<num_blocks, num_threads>>>(array, size);
  hipDeviceSynchronize();
  printf("Problem 1:");
  printArray(array, size);
  hipFree(array);

  // Problem 2: Modify the size of array to 1024

  size = 1024;
  hipMallocManaged(&array, size * sizeof(int));
  num_blocks = (size + num_threads - 1) / num_threads;
  initialize<<<num_blocks, num_threads>>>(array, size);
  hipDeviceSynchronize();
  printf("\n\nProblem 2:");
  printArray(array, size);

  // Problem 3: In problem 2 create another kernel that
  // adds i to array[i]

  addIValue<<<num_blocks, num_threads>>>(array, size);
  hipDeviceSynchronize();
  printf("\n\nProblem 3:");
  printArray(array, size);
  hipFree(array);

  // Problem 4: In problem 2, change the array size to 8000
  // Check if answer to problem 3 still works

  size = 8000; // size is changed to 8000
  hipMallocManaged(&array, size * sizeof(int));
  num_blocks = (size + num_threads - 1) / num_threads;
  addIValue<<<num_blocks, num_threads>>>(array, size);
  hipDeviceSynchronize();
  printf("\n\nProblem 4:");
  printArray(array, size);
  hipFree(array);
  hipDeviceReset();

  return 0;
}
