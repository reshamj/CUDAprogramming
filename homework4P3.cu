//Problem 3. 
//Write a CUDA program: In Problem 2, create another kernel that adds i to array[i]

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = index; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

//Kernel function to add i to a[i]
__global__
void addIValue(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x;

  for (int i = index; i < size; i+= increment){ 
         arr[i] += i; 
      }
}


void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework 4\n Problem 3:  In Problem 3, create another kernel that adds i to array[i]");

//Declare int array
int size = 1024;
int *array;
int gpuThread = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int blocks = (size + gpuThread - 1) / gpuThread;
initialize<<<blocks, gpuThread>>>(array, size);

//add value of i to array 
addIValue<<<blocks, gpuThread>>>(array, size);
hipDeviceSynchronize();

print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}
