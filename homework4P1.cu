//Problem 1. 
//Write a CUDA program to initialize an array of size 32 to all zeros in parallel.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//Kernel function to initialize array
__global__
void initialize(int *arr, int size){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = gridDim.x * blockDim.x; 

  for (int i = index; i < size; i += increment){ 
           arr[i] = 0; 
        }
}

void print(int *ar, int size){
  printf("\n");
  for (int i = 0; i < size; i++){
    printf("%d ", ar[i]);
  }
  printf("\n");
}


int main(void){
printf("Homework 4\n Problem 1: Initialize an array of size 32 to all zeros in parallel ");

//Declare int array
int size = 32;
int *array;
int gpuThread = 32;
int arraySize = size * sizeof(int);
hipMallocManaged(&array, arraySize);
int blocks = (size + gpuThread - 1) / gpuThread;
initialize<<<blocks, gpuThread>>>(array, size);

print(array, size);

hipFree(array);
hipDeviceReset();
return 0;
}
